#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "gasdev.h"

#define HandleErrorWrapper( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HandleNull( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
							waitKey();\
                            exit( EXIT_FAILURE );}}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *output_bitmap;
    float           *dev_inSrc;
    float           *dev_outSrc;
	float           *dev_Energy;
	float			*TherF3D;
	float			*Pot2;
	float			*Pot3;

	hiprandState*    devStates;
    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};


// these exist on the GPU side
texture<float>  texIn;
texture<float>  texPot2;
texture<float>  texPot3;
texture<float>  texTherF3D;
texture<float>  texOut;
texture<float>  texE;

int saveDAT, savePLT, SaveEnergy, Nthread, hNz_count;

long Nsteps, NstepsClean,  
	 Vini, Metodo, errflg, 
	 hNx, hNy, hz_AB, z_AT, 
	 hNz, hNF, hNpot, Ntotal, 
	 bytes, idum, hHold; // or int vars

float *Fini, *X, *Y, *Z, *cFinif, *hPot2, *hPot3, *Energia;

float   htau, htauP, htauC, 
		ha, hf, hfP, hv, hu, hD, hDP, 
		hDx, hDy, hB, hBP, etaI, etaR, 
		hDT, hWd, hLa, hLb, hAmpl;
		
char ext[20];
errno_t io_error;

long *hNzs;

//initialization of the parameters
const char *FILE_PARAMS_BASENAME = "Input3D.dat";
//Save RandomMatrix
char FILE_RAND_BASENAME[FILENAME_MAX];
//Save PLT File Name
char FILE_PLT_BASENAME[FILENAME_MAX];
//Save DAT File Name
char FILE_DAT_BASENAME[FILENAME_MAX];
//Open a PLT file
char FILE_OPEN_BASENAME[FILENAME_MAX];
//Open an other PLT file
char FILE_OPEN_BASENAME_2[FILENAME_MAX];
// energy file 
char FILE_E_BASENAME[FILENAME_MAX];
//Variables to open a plt
char filename [ FILENAME_MAX ];
//Save PHI total DAT File Name
char FILE_PHI_TOTAL_DAT_BASENAME[FILENAME_MAX];
//Save PHI LAYERS DAT File Name
char FILE_PHI_DAT_BASENAME_CAPAS[FILENAME_MAX];

char cero [FILENAME_MAX];
char header1 [FILENAME_MAX];
char header2 [FILENAME_MAX];
char header3 [FILENAME_MAX];

//GPU constant memory
//Matrix Size
__constant__ int Nx;
__constant__ int Ny;
__constant__ int Nz;
__constant__ int z_AB;

//On off de congelar dinamica
__constant__ int Hold;

//1 --> agrega potencial 0 lo saca
__constant__ int Npot;

// Equivalent to Temperature (C: Clean)
__constant__ float tau;
__constant__ float tauC;
__constant__ float tauP;

//Map function parameters	
__constant__ float a;
__constant__ float v;
__constant__ float u;

//compositional assymetry
__constant__ float f;
__constant__ float fP;

//Diffusion Coefficient
__constant__ float D;
__constant__ float DP;

//Diffusion Coefficient in x
__constant__ float Dx;

//Diffusion Coefficient in y
__constant__ float Dy;

//Long range interaction
__constant__ float B;
__constant__ float BP;

//Noise level
__constant__ float eta;

//Time step size
__constant__ float DT;

//Potential parameters
__constant__ float Wd;
__constant__ float La;
__constant__ float Lb;
__constant__ float Ampl;


//************************************************************************************************************
//									FUNCTIONS CUDA
//*************************************************************************************************************
__global__ void setup_kernel ( hiprandState * state, unsigned long seed );
__device__ float generate( hiprandState* globalState, int ind );
__device__ float aver3D(texture<float> texMat, int x, int y, int z);
__device__ float aver3DPot(texture<float> texMat, texture<float> texC, int x, int y, int z);
__device__ float Derivx(texture<float> texMat, int x, int y, int z) ;
__device__ float Derivy(texture<float> texMat, int x, int y, int z) ;
__device__ float grad3D(texture<float> texMat, int x, int y, int z);
__global__ void gl_kernel1(	float *TherF3D, bool ruidoSwitch, bool dstOut);
__global__ void gl_kernel2( float *Fout, hiprandState* globalState, bool ruidoSwitch, bool dstOut);
__global__ void gl_kernel4( float *Energy, bool ruidoSwitch, bool dstOut);
void make_geo();
long calc_pot2();//float escala);
long calc_pot3();

//************************************************************************************************************
//									RUNNING and MEMORY_MANAGER
//*************************************************************************************************************
long init_mem();
void run_gl( DataBlock *d);
void anim_exit( DataBlock *d );
static void HandleError( hipError_t err, const char *file, int line );

//************************************************************************************************************
//									FILE_MANAGER
//*************************************************************************************************************
errno_t get_plt(const char *fname);
errno_t get_plt_Alineado_BOTTOM(int hz_AB, const char *fname);
errno_t get_plt_Alineado_TOP(int hz_AB, const char *fname);
errno_t get_dat(char *fname);
errno_t get_param(const char *fname);
errno_t gen_rand(const char* fname, float *Fini);
errno_t gen_rand_Alineado_BOTTOM(int hz_AB, const char* fname, float *Fini);
errno_t save_plt(const char *basename, float *cFini, int q);
errno_t save_dat(const char *basename, float *cFini, int q);
errno_t save_dat_Energy(const char *basename, float EnergiaTotal, float FinalPhi, int q);
errno_t save_dat_Energy2(const char *basename, float EnergiaTotal, float FinalPhi, int k, int q);
errno_t save_dat_capas(const char *basename, float *energy, float *phi, int q);
void print_error(const char *msg, int err);
void waitKey();

//************************************************************************************************************
//									       MAIN
//*************************************************************************************************************

int main( void ) 
{
	printf("Initialization begin\n");

	//Loads the simulation parameters from file
	printf("Reading param file\n");
	if((io_error = get_param(FILE_PARAMS_BASENAME)) != 0)
		print_error("Error: opening param file", io_error);
	else
	{
		printf("OK \n");
	}
	
	printf("Verifying Nx is multiple of 16\n");
	if (!(hNx%16 == 0)) 
	{
		printf("Nx = %d is not a multiple of 16.\n", hNx); 
		getchar(); 
		exit(-1);
	}

	printf("Verifying Ny is multiple of 16\n");
	if (!(hNy%16 == 0)) 
	{
		printf("Ny = %d is not a multiple of 16\n", hNy); 
		getchar(); 
		exit(-1);
	}
	
	for(int i = 0; i < hNz_count; i++)
	{
		hNz = hNzs[i];
		
		// Calculate Ntotal
		printf("Processing Ntotal \n");
		Ntotal = hNx * hNy * hNz; 
		bytes = Ntotal*sizeof(float);

		// allocate memory for the needed arrays
		init_mem(); 
		printf("Verifying if the Potential is included..\n");
		if (hNpot == 1) 
		{
			calc_pot2();//(3.0f);
			calc_pot3();
			printf("The Potential is included..\n");
		} 
		else
		{
			printf("The Potential is NOT included..\n");
		}
		

	// if Vini = 0 I generate a random file, else I open other
	switch(Metodo)
	{
		case 0:
			if (Vini == 0)
			{
				printf("Generating random matrix...");
				if((io_error = gen_rand(FILE_RAND_BASENAME, Fini)) != 0)
				{
					print_error("Error: generating random matrix \n", io_error);
				}
				else
				{
					printf(" OK\n");
				}
			}
			break;
	
		case 1:	
			if(Vini != 0)
			{
				//// read Out file
				printf("Loading Initial file \n");
				if(strcmp(ext,"plt") == 0) 
				{
					if (Vini < 10) 
					{
						sprintf(cero, "0");
					}
					else 
					{
						sprintf(cero, "");
					}
					
					sprintf(filename, "%s%s%d.plt", FILE_OPEN_BASENAME, cero, Vini);
					printf("Loading %s... ",filename);
					errflg  =  get_plt(filename);
					if(errflg) 
					{
						print_error("Error: opening Initial .plt file", errflg);
					} 
					else 
					{
						printf("OK \n");
					}
				}

				if(strcmp(ext,"dat")==0) 
				{
					if (Vini<10) 
					{
						sprintf(cero, "0");
					} 
					else 
					{
						sprintf(cero, "");
					}
					
					sprintf(filename, "%s%s%d.dat", FILE_OPEN_BASENAME, cero, Vini);
					printf("Loading %s... ",filename);
					errflg  =  get_dat(filename);
					
					if(errflg) 
					{
						print_error("Error: opening Initial .dat file", errflg);
					} 
					else 
					{
						printf("OK \n");
					}
				}
			}
			break;
		
		case 2:
			if(Vini != 0)
				{ 
				printf("Loading Partial Initial file \n");
				if(strcmp(ext,"plt") == 0) 
				{
					if (Vini < 10) 
					{
						sprintf(cero, "0");
					}
					else 
					{
						sprintf(cero, "");
					}
					
					sprintf(filename, "%s%s%d.plt", FILE_OPEN_BASENAME, cero, Vini);
					printf("Loading %s... ",filename);
					errflg  =  get_plt_Alineado_BOTTOM(hz_AB, filename);
					
					if(errflg) 
					{
						print_error("Error: opening Initial .plt file", errflg);
					} 
					else 
					{
						printf("OK \n");
					}
				}
			

				printf("Generating random matrix...");
				if((io_error = gen_rand_Alineado_BOTTOM(hz_AB, FILE_RAND_BASENAME, Fini)) != 0)
				{
					print_error("Error: generating random matrix \n", io_error);
				}
				else
				{
					printf(" OK\n");
				}
			}
		break;	
		
		case 3:
			if(Vini != 0)
				{ 
				printf("Loading Partial Initial file \n");
				if(strcmp(ext,"plt") == 0) 
				{
					if (Vini < 10) 
					{
						sprintf(cero, "0");
					}
					else 
					{
						sprintf(cero, "");
					}
					
					sprintf(filename, "%s%s%d.plt", FILE_OPEN_BASENAME, cero, Vini);
					printf("Loading %s... ",filename);
					errflg  =  get_plt_Alineado_BOTTOM(hz_AB, filename);
			
					if(errflg) 
					{
						print_error("Error: opening Initial .plt file", errflg);
					} 
					else 
					{
						printf("OK \n");
					}
					
					sprintf(filename, "%s%s%d.plt", FILE_OPEN_BASENAME_2, cero, Vini);
					printf("Loading %s... ",filename);
					errflg  =  get_plt_Alineado_TOP(hz_AB, filename);
					
					if(errflg) 
					{
						print_error("Error: opening Initial .plt file", errflg);
					} 
					else 
					{
						printf("OK \n");
					}
				}
			}
		break;	
		
	}

		DataBlock   data;
		data.totalTime = 0;
		data.frames = 0;
		
		// random part
		if (!(etaR==0))
		{ 
			hipMalloc( &data.devStates, Ntotal*sizeof( hiprandState ) );
		}
		// end of random
		
		HandleErrorWrapper( hipEventCreate( &data.start ) );
		HandleErrorWrapper( hipEventCreate( &data.stop ) );

		HandleErrorWrapper( hipMalloc( (void**)&data.dev_inSrc, bytes ) );
		HandleErrorWrapper( hipMalloc( (void**)&data.dev_outSrc, bytes ) );
		HandleErrorWrapper( hipMalloc( (void**)&data.dev_Energy, bytes ) );
		HandleErrorWrapper( hipMalloc( (void**)&data.TherF3D, bytes ) );
		HandleErrorWrapper( hipMalloc( (void**)&data.Pot2, bytes ) ); 
		HandleErrorWrapper( hipMalloc( (void**)&data.Pot3, bytes ) ); 
		
		HandleErrorWrapper( hipBindTexture( NULL, texIn, data.dev_inSrc, bytes ) );
		HandleErrorWrapper( hipBindTexture( NULL, texOut, data.dev_outSrc, bytes ) );
		HandleErrorWrapper( hipBindTexture( NULL, texE, data.dev_Energy, bytes ) );
		HandleErrorWrapper( hipBindTexture( NULL, texTherF3D, data.TherF3D, bytes ) );
		HandleErrorWrapper( hipBindTexture( NULL, texPot2, data.Pot2, bytes ) );
		HandleErrorWrapper( hipBindTexture( NULL, texPot3, data.Pot3, bytes ) );
			
		// CUDA memory copy
		HandleErrorWrapper( hipMemcpy( data.dev_inSrc, Fini, bytes, hipMemcpyHostToDevice ) );
		HandleErrorWrapper( hipMemcpy( data.TherF3D, Fini, bytes, hipMemcpyHostToDevice ) );
		HandleErrorWrapper( hipMemcpy( data.Pot2, hPot2, bytes, hipMemcpyHostToDevice ) );
		HandleErrorWrapper( hipMemcpy( data.Pot3, hPot3, bytes, hipMemcpyHostToDevice ) );
		
		hipMemcpyToSymbol(HIP_SYMBOL(Nx), &hNx, sizeof(int) );
		hipMemcpyToSymbol(HIP_SYMBOL(Ny), &hNy, sizeof(int) );
		hipMemcpyToSymbol(HIP_SYMBOL(Nz), &hNz, sizeof(int) );
		hipMemcpyToSymbol(HIP_SYMBOL(z_AB), &hz_AB, sizeof(int) );
		
		hipMemcpyToSymbol(HIP_SYMBOL(Npot), &hNpot, sizeof(int) );
		hipMemcpyToSymbol(HIP_SYMBOL(Hold), &hHold, sizeof(int) );
		hipMemcpyToSymbol(HIP_SYMBOL(tau), &htau, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(tauP), &htauP, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(tauC), &htauC, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(a), &ha, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(f), &hf, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(fP), &hfP, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(v), &hv, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(u), &hu, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(D), &hD, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(DP), &hDP, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(Dx), &hDx, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(Dy), &hDy, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(B), &hB, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(BP), &hBP, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(eta), &etaR, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(DT), &hDT, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(Wd), &hWd, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(La), &hLa, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(Lb), &hLb, sizeof(float) );
		hipMemcpyToSymbol(HIP_SYMBOL(Ampl), &hAmpl, sizeof(float) );


		run_gl(&data);
		anim_exit( &data );
	}
	
	waitKey();
}

//************************************************************************************************************
//									FUNCTIONS CUDA
//*************************************************************************************************************

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int irand = x + y * blockDim.x * gridDim.x;
	hiprand_init ( seed, irand, 0, &state[irand] );
}

__device__ float generate( hiprandState* globalState, int ind ) 
{
    hiprandState localState = globalState[ind];
	float RANDOM = hiprand_uniform( &localState );
	float RANDOM2 = eta*(1.0f-2.0f*RANDOM);
    globalState[ind] = localState;
    return RANDOM2;
}

__device__ float aver3D(texture<float> texMat, int x, int y, int z) 
{
	float average;
	
	int ixl = x - 1;
	int ixr = x + 1;
	
	if (x == 0)
	{
		ixl=(Nx-1);
	}
	
	if (x == Nx-1) 
	{
		ixr=0; 
	}
	int iyl = y - 1;
	int iyr = y + 1;
	if (y == 0)
	{
		iyl=(Ny-1);
	}
	if (y == Ny-1) 
	{
		iyr=0; 
	}
	int izl = z - 1;
	int izr = z + 1;
	
	if (z == 0)
	{
		izl=(Nz-1);
	}
	
	if (z == Nz-1)
	{
		izr=0; 
	}
		
	float Xl, Xr, Yl, Yr, Zl, Zr, Xlt, Xrt, Xrb, Xlb, Zrl, Zrr, Zrt, Zrb, Zll, Zlr, Zlt, Zlb, Zrlt, Zrrt, Zrrb, Zrlb, Zllt, Zlrt, Zlrb, Zllb;
	
	int Nyy, Nyr, Nyl, Nzz, Nzr, Nzl;
	Nyy = y * Nx; 
	Nyr = iyr * Nx; 
	Nyl = iyl * Nx;
	Nzz = z * Nx * Ny; 
	Nzr = izr * Nx * Ny; 
	Nzl = izl * Nx * Ny;
			
	// first neighbors (6)
	Xl = tex1Dfetch(texMat,ixl + Nyy + Nzz);
	Xr = tex1Dfetch(texMat,ixr + Nyy + Nzz);
	Yl = tex1Dfetch(texMat,x + Nyl + Nzz);
	Yr = tex1Dfetch(texMat,x + Nyr + Nzz);
	Zl = tex1Dfetch(texMat,x + Nyy + Nzl);
	Zr = tex1Dfetch(texMat,x + Nyy + Nzr);
	
	// second neighbors (12)
	Xlt = tex1Dfetch(texMat,ixl + Nyr + Nzz);
	Xrt = tex1Dfetch(texMat,ixr + Nyr + Nzz);
	Xrb = tex1Dfetch(texMat,ixr + Nyl + Nzz);
	Xlb = tex1Dfetch(texMat,ixl + Nyl + Nzz);
	
	Zrl = tex1Dfetch(texMat,ixl + Nyy + Nzr);
	Zrr = tex1Dfetch(texMat,ixr + Nyy + Nzr);
	Zrt = tex1Dfetch(texMat,x + Nyr + Nzr);
	Zrb = tex1Dfetch(texMat,x + Nyl + Nzr);
	 
	Zll = tex1Dfetch(texMat,ixl + Nyy + Nzl);
	Zlr = tex1Dfetch(texMat,ixr + Nyy + Nzl);
	Zlt = tex1Dfetch(texMat,x + Nyr + Nzl);
	Zlb = tex1Dfetch(texMat,x + Nyl + Nzl);
	
	// third neighbors (8)
	Zrlt = tex1Dfetch(texMat,ixl + Nyr + Nzr);
	Zrrt = tex1Dfetch(texMat,ixr + Nyr + Nzr);
	Zrrb = tex1Dfetch(texMat,ixr + Nyl + Nzr);
	Zrlb = tex1Dfetch(texMat,ixl + Nyl + Nzr);
	
	Zllt = tex1Dfetch(texMat,ixl + Nyr + Nzl);
	Zlrt = tex1Dfetch(texMat,ixr + Nyr + Nzl);
	Zlrb = tex1Dfetch(texMat,ixr + Nyl + Nzl);
	Zllb = tex1Dfetch(texMat,ixl + Nyl + Nzl);
	
	float c_1 = 6.0f/80.0f;
	float c_2 = 3.0f/80.0f;
	float c_3 = 1.0f/80.0f;
	
	average = c_1*(Xl + Xr + Yl + Yr + Zl + Zr) + 
			  c_2*(Xlt + Xrt + Xrb + Xlb + Zrl + Zrr + Zrt + Zrb + Zll + Zlr + Zlt + Zlb) + 
			  c_3*(Zrlt + Zrrt + Zrrb + Zrlb + Zllt + Zlrt + Zlrb + Zllb);
	
	return average;
}


__device__ float aver3DPot(texture<float> texMat, texture<float> texC, int x, int y, int z) 
{
	float average;
	
	int ixl = x - 1;
	int ixr = x + 1;
	
	if (x == 0)
	{
		ixl=(Nx-1);
	}
	
	if (x == Nx-1) 
	{
		ixr=0; 
	}
	int iyl = y - 1;
	int iyr = y + 1;
	if (y == 0)    iyl=(Ny-1);
	if (y == Ny-1) iyr=0; 
	
	int izl = z - 1;
	int izr = z + 1;
	
	if (z == 0)
	{
		izl=(Nz-1);
	}
	
	if (z == Nz-1)
	{
		izr=0; 
	}
		
	float Xl, Xr, Yl, Yr, Zl, Zr, Xlt, Xrt, Xrb, Xlb, Zrl, Zrr, Zrt, Zrb, Zll, Zlr, Zlt, Zlb, Zrlt, Zrrt, Zrrb, Zrlb, Zllt, Zlrt, Zlrb, Zllb;
	
	int Nyy, Nyr, Nyl, Nzz, Nzr, Nzl;
	Nyy = y * Nx; 
	Nyr = iyr * Nx; 
	Nyl = iyl * Nx;
	Nzz = z * Nx * Ny; 
	Nzr = izr * Nx * Ny; 
	Nzl = izl * Nx * Ny;
			
	// first neighbors (6)
	Xl = tex1Dfetch(texMat,ixl + Nyy + Nzz) *  tex1Dfetch(texC,ixl + Nyy + Nzz);
	Xr = tex1Dfetch(texMat,ixr + Nyy + Nzz) *  tex1Dfetch(texC,ixr + Nyy + Nzz);
	Yl = tex1Dfetch(texMat,x   + Nyl + Nzz) *  tex1Dfetch(texC,x   + Nyl + Nzz);
	Yr = tex1Dfetch(texMat,x   + Nyr + Nzz) *  tex1Dfetch(texC,x   + Nyr + Nzz);
	Zl = tex1Dfetch(texMat,x   + Nyy + Nzl) *  tex1Dfetch(texC,x   + Nyy + Nzl);
	Zr = tex1Dfetch(texMat,x   + Nyy + Nzr) *  tex1Dfetch(texC,x   + Nyy + Nzr);
	
	// secneighbors (12)
	Xlt = tex1Dfetch(texMat,ixl + Nyr + Nzz) * 	tex1Dfetch(texC,ixl + Nyr + Nzz);
	Xrt = tex1Dfetch(texMat,ixr + Nyr + Nzz) *  tex1Dfetch(texC,ixr + Nyr + Nzz);
	Xrb = tex1Dfetch(texMat,ixr + Nyl + Nzz) *  tex1Dfetch(texC,ixr + Nyl + Nzz);
	Xlb = tex1Dfetch(texMat,ixl + Nyl + Nzz) *  tex1Dfetch(texC,ixl + Nyl + Nzz);
	
	Zrl = tex1Dfetch(texMat,ixl + Nyy + Nzr) *  tex1Dfetch(texC,ixl + Nyy + Nzr);
	Zrr = tex1Dfetch(texMat,ixr + Nyy + Nzr) *  tex1Dfetch(texC,ixr + Nyy + Nzr);
	Zrt = tex1Dfetch(texMat,x   + Nyr + Nzr) *  tex1Dfetch(texC,x   + Nyr + Nzr);;
	Zrb = tex1Dfetch(texMat,x   + Nyl + Nzr) *  tex1Dfetch(texC,x   + Nyl + Nzr);
	 	  
	Zll = tex1Dfetch(texMat,ixl + Nyy + Nzl) *  tex1Dfetch(texC,ixl + Nyy + Nzl);
	Zlr = tex1Dfetch(texMat,ixr + Nyy + Nzl) *  tex1Dfetch(texC,ixr + Nyy + Nzl);
	Zlt = tex1Dfetch(texMat,x   + Nyr + Nzl) *  tex1Dfetch(texC,x   + Nyr + Nzl);;
	Zlb = tex1Dfetch(texMat,x   + Nyl + Nzl) *  tex1Dfetch(texC,x   + Nyl + Nzl);
	
		 
	// thieighbors (8)
	Zrlt = tex1Dfetch(texMat,ixl + Nyr + Nzr) * tex1Dfetch(texC,ixl + Nyr + Nzr);
	Zrrt = tex1Dfetch(texMat,ixr + Nyr + Nzr) * tex1Dfetch(texC,ixr + Nyr + Nzr);
	Zrrb = tex1Dfetch(texMat,ixr + Nyl + Nzr) * tex1Dfetch(texC,ixr + Nyl + Nzr);
	Zrlb = tex1Dfetch(texMat,ixl + Nyl + Nzr) * tex1Dfetch(texC,ixl + Nyl + Nzr);
		  													  
	Zllt = tex1Dfetch(texMat,ixl + Nyr + Nzl) * tex1Dfetch(texC,ixl + Nyr + Nzl);
	Zlrt = tex1Dfetch(texMat,ixr + Nyr + Nzl) * tex1Dfetch(texC,ixr + Nyr + Nzl);
	Zlrb = tex1Dfetch(texMat,ixr + Nyl + Nzl) * tex1Dfetch(texC,ixr + Nyl + Nzl);
	Zllb = tex1Dfetch(texMat,ixl + Nyl + Nzl) * tex1Dfetch(texC,ixl + Nyl + Nzl);

	
	float c_1 = 6.0f/80.0f;
	float c_2 = 3.0f/80.0f;
	float c_3 = 1.0f/80.0f;
	
	average =    c_1*(Xl + Xr + Yl + Yr + Zl + Zr) + 
		    	 c_2*(Xlt + Xrt + Xrb + Xlb + Zrl + Zrr + Zrt + Zrb + Zll + Zlr + Zlt + Zlb) + 
				 c_3*(Zrlt + Zrrt + Zrrb + Zrlb + Zllt + Zlrt + Zlrb + Zllb);
	
	return average;
	
}


__device__ float grad3D(texture<float> texMat, int x, int y, int z) 
{
	float gradient, gradient2, dx, dy, dz, c, delta;
	
	int ixl = x - 1;
	int ixr = x + 1;
	
	if (x == 0)
	{
		ixl=(Nx-1);
	}
	
	if (x == Nx-1) 
	{
		ixr=0; 
	}
	int iyl = y - 1;
	int iyr = y + 1;
	if (y == 0)
	{
		iyl=(Ny-1);
	}
	if (y == Ny-1) 
	{
		iyr=0; 
	}
	int izl = z - 1;
	int izr = z + 1;
	
	if (z == 0)
	{
		izl=(Nz-1);
	}
	
	if (z == Nz-1)
	{
		izr=0; 
	}


	float Xr, Yr, Zr, Xl, Yl, Zl, Xlt;
	
	int Nyy, Nyr, Nyl, Nzz, Nzr, Nzl;
	
	delta = powf((40.0f/11.0f),(0.5f));

	Nyy = y * Nx; 
	Nyr = iyr * Nx; 
	Nyl = iyl * Nx;
	Nzz = z * Nx * Ny; 
	Nzr = izr * Nx * Ny; 
	Nzl = izl * Nx * Ny;
	
	c = tex1Dfetch(texMat, x + Nyy + Nzz);
	
	
	Xl = tex1Dfetch(texMat,ixl + Nyy + Nzz);
	Xr = tex1Dfetch(texMat,ixr + Nyy + Nzz);
	
	Yl = tex1Dfetch(texMat,x + Nyl + Nzz);
	Yr = tex1Dfetch(texMat,x + Nyr + Nzz);
	
	Zl = tex1Dfetch(texMat,x + Nyy + Nzl);
	Zr = tex1Dfetch(texMat,x + Nyy + Nzr);

	
	dx = (Xr - Xl)/(2.0f*delta);
	dy = (Yr - Yl)/(2.0f*delta);
	dz = (Zr - Zl)/(2.0f*delta);
	
	gradient2 = powf(dx,2) + powf(dy,2) + powf(dz,2);
	
	gradient =  powf(gradient2,0.5f);
	
	return gradient;
	
}

__device__ float Derivx(texture<float> texMat, int x, int y, int z) 
{
	float d2x;
	
	int ixl = x - 1;
	int ixr = x + 1;
	
	if (x == 0)
	{
		ixl=(Nx-1);
	}
		if (x == Nx-1) 
	{
		ixr=0; 
	}
			
	float Xl, Xr;
	float c_1 = 1.0f/2.0f;
	int Nyy, Nzz;
	
	Nyy = y * Nx; 
	Nzz = z * Nx * Ny; 
					
	Xl = tex1Dfetch(texMat,ixl + Nyy + Nzz);
	Xr = tex1Dfetch(texMat,ixr + Nyy + Nzz);
		
	d2x = c_1*(Xr + Xl) ;
	
	return d2x;
}

__device__ float Derivy(texture<float> texMat, int x, int y, int z) 
{
	float d2y;
	
	int iyl = y - 1;
	int iyr = y + 1;
	
	if (y == 0)
	{
		iyl=(Ny-1);
	}
	if (y == Ny-1)
	{
		iyr=0; 
	}
	
		
	float Yl, Yr;
	
	int Nyr, Nyl, Nzz;
	
	Nyr = iyr * Nx; 
	Nyl = iyl * Nx;
	Nzz = z * Nx * Ny; 
	
			
	Yl = tex1Dfetch(texMat,x + Nyl + Nzz);
	Yr = tex1Dfetch(texMat,x + Nyr + Nzz);
	
	float c_1 = 1.0f/2.0f;
	
	d2y = c_1*(Yr + Yl);
	
	return d2y;
	
}

__global__ void gl_kernel1(	float *TherF3D, bool ruidoSwitch, bool dstOut)
{
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;
	
	int i = x + y * Nx + z * Nx * Ny;
	
	float c, average, tauL, d2x, d2y;
			
	if (x < Nx && y < Ny && z < Nz) 
	{
		//ENTRO SI: hold es 0 entro//  hold es 1 y z mayor a z_AB 
		
		switch(Hold)
		{
			case 0: //MONOCAPA
				
				float c_1 = 1.0f - 2.0f*f;
				if (dstOut) 
				{
					c = tex1Dfetch(texIn, i); // center value of Fini
					average = aver3D(texIn, x, y, z);
					d2x = Derivx(texIn, x, y, z);
					d2y = Derivy(texIn, x, y, z);
				} 
				else 
				{// center value of Fini
					c = tex1Dfetch(texOut, i); 
					average = aver3D(texOut, x, y, z);
					d2x = Derivx(texOut, x, y, z);
					d2y = Derivy(texOut, x, y, z);
				}
			
			  
				//// Solving Ginzburg-Landau Equation
				//TherF3D // TherF3D=Fm-D*(Aver3D(Fini)-Fini) // Aver3D := laplacian // Fm=MapF3D(Fini)
											
				TherF3D[i] = -(tau - a * powf(c_1,2)) * c
									+ v * c_1 *powf(c,2)
									+ u * powf(c,3)  
									- D * (average - c) 
									- Dx * (d2x - c) 
									- Dy * (d2y - c) ;



			case 1: // BIACPA: PARTE DE ABAJO ES FIJA

				if (z > z_AB)
				{
					float c_1 = 1.0f - 2.0f*fP;

					if (dstOut) 
					{
						c = tex1Dfetch(texIn, i); // center value of Fini
						average = aver3D(texIn, x, y, z);
						d2x = Derivx(texIn, x, y, z);
						d2y = Derivy(texIn, x, y, z);
					} 
					else 
					{
						c = tex1Dfetch(texOut, i); // center value of Fini
						average = aver3D(texOut, x, y, z);
						d2x = Derivx(texOut, x, y, z);
						d2y = Derivy(texOut, x, y, z);
					}
			
			//// Solving Ginzburg-Landau Equation
			//TherF3D // TherF3D=Fm-D*(Aver3D(Fini)-Fini) // Aver3D := laplacian // Fm=MapF3D(Fini)
			
								
					TherF3D[i]=-(tauP - a * powf(c_1,2)) * c
										  +	v * c_1 *powf(c,2)
										  +	u * powf(c,3)  
										  -	(DP) * (average - c) 
										  - Dx * (d2x - c) 
										  - Dy * (d2y - c) ;							
				}

				else
				{
					float c_1 = 1.0f - 2.0f*f;
					if (dstOut) 
					{
						c = tex1Dfetch(texIn, i); // center value of Fini
					} 
					else 
					{// center value of Fini
						c = tex1Dfetch(texOut, i); 						
					}
				
						TherF3D[i] = c;		//	 ABAJO FIJO ES SOLO IGUAL A C		
				}


				case 2: // BIACPA: PARTE DE ABAJO EVOLUCIONA TAMBIEN
					
				if (z > z_AB)
				{
					float c_1 = 1.0f - 2.0f*fP;

					if (dstOut) 
					{
						c = tex1Dfetch(texIn, i); // center value of Fini
						average = aver3D(texIn, x, y, z);
						d2x = Derivx(texIn, x, y, z);
						d2y = Derivy(texIn, x, y, z);
					} 
					else 
					{
						c = tex1Dfetch(texOut, i); // center value of Fini
						average = aver3D(texOut, x, y, z);
						d2x = Derivx(texOut, x, y, z);
						d2y = Derivy(texOut, x, y, z);
					}
			
			//// Solving Ginzburg-Landau Equation
			//TherF3D // TherF3D=Fm-D*(Aver3D(Fini)-Fini) // Aver3D := laplacian // Fm=MapF3D(Fini)
			
								
						TherF3D[i]=-(tauP - a * powf(c_1,2)) * c
										  +	v * c_1 *powf(c,2)
										  +	u * powf(c,3)  
										  -	(DP) * (average - c) 
										  - Dx * (d2x - c) 
										  - Dy * (d2y - c) ;
								
				}
				else
				{
					float c_1 = 1.0f - 2.0f*f;
					if (dstOut) 
					{
						c = tex1Dfetch(texIn, i); // center value of Fini
						average = aver3D(texIn, x, y, z);
						d2x = Derivx(texIn, x, y, z);
						d2y = Derivy(texIn, x, y, z);
					} 
					else 
					{// center value of Fini
						c = tex1Dfetch(texOut, i); 
						average = aver3D(texOut, x, y, z);
						d2x = Derivx(texOut, x, y, z);
						d2y = Derivy(texOut, x, y, z);
					}
				
			
			  
					//// Solving Ginzburg-Landau Equation
					//TherF3D // TherF3D=Fm-D*(Aver3D(Fini)-Fini) // Aver3D := laplacian // Fm=MapF3D(Fini)
											
						TherF3D[i] = -(tau - a * powf(c_1,2)) * c
										+ v * c_1 *powf(c,2)
										+ u * powf(c,3)  
										- D * (average - c) 
										- Dx * (d2x - c) 
										- Dy * (d2y - c) ;
				}
		}
	}
} 

__global__ void gl_kernel2( float *Fout, hiprandState* globalState, bool ruidoSwitch, bool dstOut) 
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;
	
	int irand = x + y * blockDim.x * gridDim.x;
	
	int i = x + y * Nx + z * Nx * Ny;
	
	float c, TF, TFav, CenterPot, CenterPot2, AverPot, AverPot2;
		
	
	if (x < Nx && y < Ny && z < Nz) 
	{			
		switch(Hold)
		{
			case 0: //MONOCAPA

				if (dstOut) 
				{
					c = tex1Dfetch(texIn, i); 
					AverPot = aver3DPot(texPot2, texIn, x, y, z); 
				
				} 
				else 
				{
					c = tex1Dfetch(texOut, i);
					AverPot = aver3DPot(texPot2, texOut, x, y, z); 
				}	

				CenterPot = tex1Dfetch(texPot2, i);// center value of texPot2c

				TF = tex1Dfetch(texTherF3D, i); 

				TFav = aver3D(texTherF3D, x, y, z);
		   
				AverPot2 = aver3D(texPot3, x, y, z);
				CenterPot2 = tex1Dfetch(texPot3, i);// center value of texPot2c

				float noise = 0.0f;
				if(ruidoSwitch) 
				{
					if (!(eta==0.0f)) 
					{
						noise=generate(globalState, irand);
					}
				}
				if (Npot) 
				{ 
					Fout[i] = c + DT * (TFav - TF - B * c) +  DT * (AverPot - CenterPot * c ) + DT * (AverPot2 - CenterPot2) +  noise ;
    			}
				else
				{ 	
					Fout[i] = c + DT * (TFav - TF - (B) * c)  +   noise ;
				}


			case 1: // BIACPA: PARTE DE ABAJO ES FIJA

			if (z > z_AB)
			{
			
				if (dstOut) 
				{
					c = tex1Dfetch(texIn, i); 
					AverPot = aver3DPot(texPot2, texIn, x, y, z); 
				} 
				else 
				{
					c = tex1Dfetch(texOut, i);
					AverPot = aver3DPot(texPot2, texOut, x, y, z); 
				}	

				TF = tex1Dfetch(texTherF3D, i); 

				TFav = aver3D(texTherF3D, x, y, z);

				CenterPot = tex1Dfetch(texPot2, i); // center value of texPot2
			
				AverPot2 = aver3D(texPot3, x, y, z);
				CenterPot2 = tex1Dfetch(texPot3, i); // center value of texPot3
				
			
				float noise = 0.0f;
				if(ruidoSwitch) 
				{
					if (!(eta==0.0f)) 
					{
						noise=generate(globalState, irand);
					}
				}
				
				if (Npot) 
				{ 
					Fout[i] = c + DT * (TFav - TF - (BP) * c)  +  DT * (AverPot - CenterPot * c ) + DT * (AverPot2 - CenterPot2) + noise;
				}
				else
				{ 
					Fout[i] = c + DT * (TFav - TF - (BP) * c)  +  noise;
				}

			}

			else
			{
				if (dstOut) 
				{
					c = tex1Dfetch(texIn, i); 
					AverPot = aver3DPot(texPot2, texIn, x, y, z); 
				
				} 
				else 
				{
					c = tex1Dfetch(texOut, i);
					AverPot = aver3DPot(texPot2, texOut, x, y, z); 
				}	

				CenterPot = tex1Dfetch(texPot2, i);// center value of texPot2c  
				AverPot2 = aver3D(texPot3, x, y, z);
				CenterPot2 = tex1Dfetch(texPot3, i);// center value of texPot2c

				float noise = 0.0f;
				if(ruidoSwitch) 
				{
					if (!(eta==0.0f)) 
					{
						noise=generate(globalState, irand);
					}
				}
				if (Npot) 
				{ 
						 Fout[i] = c + DT * (AverPot - CenterPot * c ) + DT * (AverPot2 - CenterPot2);  //ABAJO FIJO ES ESTO
				}
				else
				{ 	
						Fout[i] = c ;// ABAJO FIJO ES ESTO
				}
		}


			case 2: // BIACPA: PARTE DE ABAJO TIENE DINAMICA

			if (z > z_AB)
			{
			
				if (dstOut) 
				{
					c = tex1Dfetch(texIn, i); 
					AverPot = aver3DPot(texPot2, texIn, x, y, z); 
				} 
				else 
				{
					c = tex1Dfetch(texOut, i);
					AverPot = aver3DPot(texPot2, texOut, x, y, z); 
				}	

				TF = tex1Dfetch(texTherF3D, i); 

				TFav = aver3D(texTherF3D, x, y, z);

				CenterPot = tex1Dfetch(texPot2, i); // center value of texPot2
				AverPot2 = aver3D(texPot3, x, y, z);
				CenterPot2 = tex1Dfetch(texPot3, i); // center value of texPot3
				
			
				float noise = 0.0f;
				if(ruidoSwitch) 
				{
					if (!(eta==0.0f)) 
					{
						noise=generate(globalState, irand);
					}
				}
				
				if (Npot) 
				{ 
					Fout[i] = c + DT * (TFav - TF - (BP) * c)  +  DT * (AverPot - CenterPot * c ) + DT * (AverPot2 - CenterPot2) + noise;
				}
				else
				{ 
					Fout[i] = c + DT * (TFav - TF - (BP) * c)  +  noise;
				}

			}

			else
			{
				if (dstOut) 
				{
					c = tex1Dfetch(texIn, i); 
					AverPot = aver3DPot(texPot2, texIn, x, y, z); 
				
				} 
				else 
				{
					c = tex1Dfetch(texOut, i);
					AverPot = aver3DPot(texPot2, texOut, x, y, z); 
				}	

				CenterPot = tex1Dfetch(texPot2, i);// center value of texPot2c

				TF = tex1Dfetch(texTherF3D, i); 

				TFav = aver3D(texTherF3D, x, y, z);
		   
				AverPot2 = aver3D(texPot3, x, y, z);
				CenterPot2 = tex1Dfetch(texPot3, i);// center value of texPot2c

				float noise = 0.0f;
				if(ruidoSwitch) 
				{
					if (!(eta==0.0f)) 
					{
						noise=generate(globalState, irand);
					}
				}
				if (Npot) 
				{ 
					Fout[i] = c + DT * (TFav - TF - B * c) +  DT * (AverPot - CenterPot * c ) + DT * (AverPot2 - CenterPot2) +  noise ;
    			}
				else
				{ 	
					Fout[i] = c + DT * (TFav - TF - (B) * c)  +   noise ;
				}
			}
		}
	}
}

			
__global__ void gl_kernel4( float *Energy, bool ruidoSwitch, bool dstOut) 
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;
	
	int irand = x + y * blockDim.x * gridDim.x;
	
	int i = x + y * Nx + z * Nx * Ny;
	
	float tauL, CenterLaplacian3, AverLaplacian3, GradLaplacian3, Laplaciano;

				
	if (x<Nx && y<Ny && z<Nz) 
	{	

		switch(Hold)
		{
			case 0: //MONOCAPA
				
				float c_1 = 0.5f*(powf(D,3/2)/powf(B,0.5f));
				float c_2 = - D;
				float c_3 = (0.5f)*(-tau + a*powf((1.0f-2.0f*f),2) + 3 * powf(B*D, 0.5f)  );
				float c_4 = (v/3)*(1.0f-2.0f*fP);
				float c_5 = (u/4);

				if (dstOut) 
				{
					CenterLaplacian3 = tex1Dfetch(texIn, i); 
					AverLaplacian3 = aver3D(texIn, x, y, z);
					GradLaplacian3 = grad3D(texIn, x, y, z);
				} 
				else 
				{
					CenterLaplacian3 = tex1Dfetch(texOut, i); 
					AverLaplacian3 = aver3D(texOut, x, y, z);
					GradLaplacian3 = grad3D(texOut, x, y, z);
				}
				
				if (ruidoSwitch) 
				{
					tauL = tau; 
				} 
				else 
				{
					tauL = tauC;
				}
				
				Laplaciano = AverLaplacian3 - CenterLaplacian3;
			
				Energy[i] =     	c_1 * powf(Laplaciano,2) +
									c_2 * powf(GradLaplacian3,2) + 
									c_3 * powf(CenterLaplacian3,2) + 
									c_4 * powf(CenterLaplacian3,3) ;//+
									c_5 * powf(CenterLaplacian3,4);		
		
			case 1: // BIACPA: PARTE DE ABAJO ES FIJA

				if (z > z_AB)
				{
					float c_1 = 0.5f*(powf(DP,3/2)/powf(BP,0.5f));
					float c_2 = - DP;
					float c_3 = (0.5f)*(-tauP + a*powf((1.0f-2.0f*fP),2) + 3 * powf(BP*DP, 0.5f)  );
					float c_4 = (v/3)*(1.0f-2.0f*fP);
					float c_5 = (u/4);

					if (dstOut) 
					{
						CenterLaplacian3 = tex1Dfetch(texIn, i); 
						AverLaplacian3 = aver3D(texIn, x, y, z);
						GradLaplacian3 = grad3D(texIn, x, y, z);
					} 
					else 
					{
						CenterLaplacian3 = tex1Dfetch(texOut, i); 
						AverLaplacian3 = aver3D(texOut, x, y, z);
						GradLaplacian3 = grad3D(texOut, x, y, z);
					}
				
					if (ruidoSwitch) 
					{
						tauL = tau; 
					} 
					else 
					{
						tauL = tauC;
					}
				
						Laplaciano = AverLaplacian3 - CenterLaplacian3;
			
						Energy[i] =     	c_1 * powf(Laplaciano,2) +
											c_2 * powf(GradLaplacian3,2) + 
											c_3 * powf(CenterLaplacian3,2) + 
											c_4 * powf(CenterLaplacian3,3);// +
											c_5 * powf(CenterLaplacian3,4);		
				}
		
				else
				{

					float c_1 = 0.5f*(powf(D,3/2)/powf(B,0.5f));
					float c_2 = - D;
					float c_3 = (0.5f)*(-tau + a*powf((1.0f-2.0f*f),2) + 3 * powf(B*D, 0.5f)  );
					float c_4 = (v/3)*(1.0f-2.0f*fP);
					float c_5 = (u/4);

					if (dstOut) 
					{
						CenterLaplacian3 = tex1Dfetch(texIn, i); 
						AverLaplacian3 = aver3D(texIn, x, y, z);
						GradLaplacian3 = grad3D(texIn, x, y, z);
					} 
					else 
					{
						CenterLaplacian3 = tex1Dfetch(texOut, i); 
						AverLaplacian3 = aver3D(texOut, x, y, z);
						GradLaplacian3 = grad3D(texOut, x, y, z);
					}
				
					if (ruidoSwitch) 
					{
						tauL = tau; 
					} 
					else 
					{
						tauL = tauC;
					}
				
					Laplaciano = AverLaplacian3 - CenterLaplacian3;
			
					Energy[i] =     	c_1 * powf(Laplaciano,2) +
										c_2 * powf(GradLaplacian3,2) + 
										c_3 * powf(CenterLaplacian3,2) + 
										c_4 * powf(CenterLaplacian3,3) +
										c_5 * powf(CenterLaplacian3,4);		
				}

			case 2: // BIACPA: PARTE DE ABAJO TIENE DINAMICA

				if (z > z_AB)
				{
					float c_1 = 0.5f*(powf(DP,3/2)/powf(BP,0.5f));
					float c_2 = - DP;
					float c_3 = (0.5f)*(-tauP + a*powf((1.0f-2.0f*fP),2) + 3 * powf(BP*DP, 0.5f)  );
					float c_4 = (v/3)*(1.0f-2.0f*fP);
					float c_5 = (u/4);

					if (dstOut) 
					{
						CenterLaplacian3 = tex1Dfetch(texIn, i); 
						AverLaplacian3 = aver3D(texIn, x, y, z);
						GradLaplacian3 = grad3D(texIn, x, y, z);
					} 
					else 
					{
						CenterLaplacian3 = tex1Dfetch(texOut, i); 
						AverLaplacian3 = aver3D(texOut, x, y, z);
						GradLaplacian3 = grad3D(texOut, x, y, z);
					}
				
					if (ruidoSwitch) 
					{
						tauL = tau; 
					} 
					else 
					{
						tauL = tauC;
					}
				
						Laplaciano = AverLaplacian3 - CenterLaplacian3;
			
						Energy[i] =     	c_1 * powf(Laplaciano,2) +
											c_2 * powf(GradLaplacian3,2) + 
											c_3 * powf(CenterLaplacian3,2) + //este hay q sacar
											c_4 * powf(CenterLaplacian3,3);// +
											c_5 * powf(CenterLaplacian3,4);		
				}
		
				else
				{

					float c_1 = 0.5f*(powf(D,3/2)/powf(B,0.5f));
					float c_2 = - D;
					float c_3 = -(0.5f)*(tau - a*powf((1.0f-2.0f*f),2) + 3 * powf(B*D, 0.5f)  );
					float c_4 = (v/3)*(1.0f-2.0f*fP);
					float c_5 = (u/4);

					if (dstOut) 
					{
						CenterLaplacian3 = tex1Dfetch(texIn, i); 
						AverLaplacian3 = aver3D(texIn, x, y, z);
						GradLaplacian3 = grad3D(texIn, x, y, z);
					} 
					else 
					{
						CenterLaplacian3 = tex1Dfetch(texOut, i); 
						AverLaplacian3 = aver3D(texOut, x, y, z);
						GradLaplacian3 = grad3D(texOut, x, y, z);
					}
				
					if (ruidoSwitch) 
					{
						tauL = tau; 
					} 
					else 
					{
						tauL = tauC;
					}
				
					Laplaciano = AverLaplacian3 - CenterLaplacian3;
			
					Energy[i] =     	c_1 * powf(Laplaciano,2) +
										c_2 * powf(GradLaplacian3,2) + 
										c_3 * powf(CenterLaplacian3,2) + //este hay q sacar
										c_4 * powf(CenterLaplacian3,3) ;//+
										c_5 * powf(CenterLaplacian3,4);		
				}
		}
	}
}


long calc_pot2()//(float escala)
{
	int Xv, Yv, Zv;
	Xv = 0; Yv = 0; Zv = 0;

 	for (int i = 0; i < Ntotal; i++) 
	{
		
		float c_1 = hAmpl/2.0f;
		
		hPot2[i] = c_1 * (tanh(-((float)Zv - hLb) * hWd) -
					      tanh(-((float)Zv - hLa) * hWd)) + hAmpl;

		//	if(Zv <= hNz / 2)
		//{   
		//	float pot = powf(sinf((float)Zv  / escala ), 2);
		//	hPot2[i] += c_1 * ( fmin(pot, 0.4f) - 0.4f );
		//}

		/*					
		hPot2[i] = hAmpl/2.0f*(tanh(-(((float)Zv-1.0f)-hLb)*hWd) -
							tanh(-(((float)Zv-1.0f)-hLa)*hWd));
		*/									
		Xv = Xv + 1;
		if(Xv == hNx) 
		{
			Xv = 0;
			Yv = Yv+1;
			if(Yv == hNy) 
			{
				Yv = 0;
				Zv = Zv+1;
			}
		}
	
	}
	
	//save_plt(hPot2,30);
	
 return(0);
}


long calc_pot3()//(float escala)
{
	int Xv, Yv, Zv;
	Xv = 0; Yv = 0; Zv = 0;

 	for (int i = 0; i < Ntotal; i++) 
	{
		
		float c_1 = - hAmpl/2.0f;
		
		hPot3[i] = (c_1/10) * (tanh(-((float)Zv - hLb) * hWd) -
					      tanh(-((float)Zv - hLa) * hWd)) + hAmpl;

	//	if(Zv <= hNz / 2)
	//{   
	//	float pot = powf(sinf((float)Zv  / escala ), 2);
	//	hPot2[i] += c_1 * ( fmin(pot, 0.4f) - 0.4f );
	//}

		/*					
		hPot2[i] = hAmpl/2.0f*(tanh(-(((float)Zv-1.0f)-hLb)*hWd) -
							tanh(-(((float)Zv-1.0f)-hLa)*hWd));
		*/									
		Xv = Xv + 1;
		if(Xv == hNx) 
		{
			Xv = 0;
			Yv = Yv+1;
			if(Yv == hNy) 
			{
				Yv = 0;
				Zv = Zv+1;
			}
		}
	
	}
	
	//save_plt(hPot2,30);
	
 return(0);
}

//////// making geometry of system
void make_geo()
{

	// for (int i=0; i<=N-1; i++) {
		// for (int j=0; j<=N-1; j++) {
			// for (int k=0; j<=N-1; k++) {
		
		  	// X[i]=(i+1)*hL/N;
			// Y[i]=(j+1)*hL/N;
			// Z[i]=hAmp*cos(PI*nx*(i+1)/N)*cos(PI*my*(j+1)/N);
			
			// }
	
		// }
	// } 
  return;
}

//************************************************************************************************************
//									MEMORY MANAGMENT
//*************************************************************************************************************

// allocate memory for the needed arrays
long init_mem()
{
 Fini = (float *) calloc((size_t) Ntotal, sizeof(float) );
 cFinif = (float *) calloc((size_t) Ntotal, sizeof(float) );
 Energia = (float *) calloc((size_t) Ntotal, sizeof(float) );
 X = (float *) calloc((size_t) Ntotal, sizeof(float) );
 Y = (float *) calloc((size_t) Ntotal, sizeof(float) );
 Z = (float *) calloc((size_t) Ntotal, sizeof(float) );
 hPot2 = (float *) calloc((size_t) Ntotal, sizeof(float) );
 hPot3 = (float *) calloc((size_t) Ntotal, sizeof(float) );

 // ==============================================================
 if(Fini == NULL || cFinif==NULL || X==NULL || Y==NULL || Z==NULL || hPot2==NULL || Energia==NULL)
 {
	 printf("ERROR while allocating arrays \n");
	 waitKey();
	 exit(-1);
 }

 return 0;
}

//Handle the errors
static void HandleError( hipError_t err, const char *file, int line ) 
{
    if (err != hipSuccess) 
	{
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		waitKey();
        exit( err );
    }
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d ) 
{
	printf( "Ending program... \n"); 
    hipUnbindTexture( texIn );
	hipUnbindTexture( texPot2 );
	hipUnbindTexture( texPot3 );
	hipUnbindTexture( texTherF3D );
    hipUnbindTexture( texOut );
	hipUnbindTexture( texE );
	
    HandleErrorWrapper( hipFree( d->dev_inSrc ) );
    HandleErrorWrapper( hipFree( d->dev_outSrc ) );
	HandleErrorWrapper( hipFree( d->dev_Energy ) );
	HandleErrorWrapper( hipFree( d->Pot2 ) );
	HandleErrorWrapper( hipFree( d->TherF3D ) );

    HandleErrorWrapper( hipEventDestroy( d->start ) );
    HandleErrorWrapper( hipEventDestroy( d->stop ) );
	printf( "End \n"); 
}

void run_gl( DataBlock *d) {
	dim3 blocks(hNx/Nthread, hNy/Nthread, hNz);
	dim3 threads(Nthread, Nthread);
	
	// Setting up cuRand in GPU
	if (!(etaR==0.0f)) 
	{
		printf( "Including noise..."); 
		//printf( "Setting up cuRand in GPU... "); 
		setup_kernel<<<blocks,threads>>>( d->devStates, idum ); 
		printf( "OK \n");
	}
	else
	{
		printf( "Noise is NOT included \n");
		//printf( "ERROR setting up cuRand in GPU... "); 
	}
	
	for (int k=0; k<hNF; k++) 
	{
		HandleErrorWrapper( hipEventRecord( d->start, 0 ) );
		
		// since tex is global and bound, we have to use ha flag to
		// select which is in/out per iteration
		volatile bool dstOut = true;
		volatile bool ruidoSwitch = true;
	
		float   *out, *PhiCuadrado, *FinalPhi2, *E, *EnergiaTotal;
	
		EnergiaTotal = (float *) calloc((size_t) hNz, sizeof(float) );
		PhiCuadrado = (float *) calloc((size_t) hNz, sizeof(float) );
		FinalPhi2 = (float *) calloc((size_t) hNz, sizeof(float) );
	
		int q = d->frames+Vini+1;
	
		float FinalPhi;
		float FinalEnergy;
		
		printf( "Running... \n"); 
		for (int i=0; i<Nsteps; i++) 
		{ 
			if (dstOut) 
			{
				out = d->dev_outSrc;
			} 
			else 
			{
				out = d->dev_inSrc;
			}

			HandleErrorWrapper( hipMemcpy( Fini, d->TherF3D, bytes, hipMemcpyDeviceToHost ) );

			gl_kernel1<<<blocks,threads>>>(d->TherF3D, ruidoSwitch, dstOut);

			HandleErrorWrapper( hipMemcpy( Fini, d->TherF3D, bytes, hipMemcpyDeviceToHost ) );

			gl_kernel2<<<blocks,threads>>>(out ,d->devStates, ruidoSwitch, dstOut);

			HandleErrorWrapper( hipMemcpy( Fini, out, bytes, hipMemcpyDeviceToHost ) );

			dstOut = !dstOut;

			//************************************************************************************************************
			//									FOR N STEPS
			//*************************************************************************************************************
			int Nsave = fmax(1.0, Nsteps / 1000);
			if (i % Nsave == 0)
			{	
				HandleErrorWrapper( hipMemcpy( cFinif, d->dev_inSrc, bytes, hipMemcpyDeviceToHost ) );
				
				E = d->dev_Energy;
			
				gl_kernel4<<<blocks,threads>>>(E, ruidoSwitch, !dstOut);
			
				HandleErrorWrapper( hipMemcpy( Energia, d->dev_Energy, bytes, hipMemcpyDeviceToHost ) );	
					
				for(int p = 0; p < hNz; p++)
				{
					PhiCuadrado[p] = 0;
					FinalPhi2[p] = 0;
					EnergiaTotal[p] = 0;
				}
				
				float FinalPhi3  = 0.0f;
				float EnergiaTotal2 = 0.0f;
				int hNxy = hNx * hNy;
				
				for(int j = 0; j < hNz; j++)
				{		
					for(int l = 0; l < hNxy; l++)
					{				
						PhiCuadrado[j] = PhiCuadrado[j] + cFinif[ j * hNxy + l ] * cFinif[ j * hNxy + l ]; //PHI^2 BY LAYERS
						
						EnergiaTotal[j] = EnergiaTotal[j] + Energia[ j * hNxy + l ]; //saving energy by layers
					}
					
					FinalPhi2[j] = sqrtf(PhiCuadrado[j] / hNxy); //saving phi by layers
 
					FinalPhi3 = FinalPhi3 + FinalPhi2[j];//integral of phi
										
					EnergiaTotal2 = EnergiaTotal2 + (EnergiaTotal[j] / hNxy);//integral of energy
				}
			 
				FinalPhi = 0.0f;
				FinalEnergy = 0.0f;	
							
				FinalPhi = (FinalPhi3 / hNz);//Normalizing
				FinalEnergy = (EnergiaTotal2 / hNz);//Normalizing
						
				int time = 0;
				
				time = i + k * Nsteps;	
								
				if((io_error = save_dat_Energy2(FILE_PHI_TOTAL_DAT_BASENAME, FinalEnergy, FinalPhi, time, q)) != 0)
				{
					print_error("Error: saving .dat file", io_error);
				}
			}
						
			//************************************************************************************************************
			//									END BY STEPS
			//*************************************************************************************************************
		}
		
		if (etaR!=0.0f && NstepsClean>0) 
		{
			ruidoSwitch = false;
			printf( "Cleaning noise... ");
			for (int i = 0; i < NstepsClean; i++) 
			{ 
				if (dstOut) 
				{
					out = d->dev_outSrc;
				} 
				else 
				{
					out = d->dev_inSrc;
				}
				
				gl_kernel1<<<blocks,threads>>>(d->TherF3D,ruidoSwitch,dstOut);
				gl_kernel2<<<blocks,threads>>>(out,d->devStates,ruidoSwitch,dstOut);
				dstOut = !dstOut;
			}

		printf( "OK \n ");
		}
		
		//Data: device to host
		HandleErrorWrapper( hipMemcpy( cFinif, d->dev_inSrc, bytes, hipMemcpyDeviceToHost ) );	
		HandleErrorWrapper( hipMemcpy( Energia, d->dev_Energy, bytes, hipMemcpyDeviceToHost ) );				
		HandleErrorWrapper( hipEventRecord( d->stop, 0 ) );
		HandleErrorWrapper( hipEventSynchronize( d->stop ) );
		
		float   elapsedTime;
		HandleErrorWrapper( hipEventElapsedTime( &elapsedTime, d->start, d->stop ) );
		d->totalTime += elapsedTime;
		++d->frames;

		printf( "File %d of %d ...",q , hNF+Vini);
		printf( "Time: %7.4f seg.\n", d->totalTime/(1000*d->frames)  );
		

		//Data saving
		printf("Saving data to filesystem. Step: %i\n", q);
		if (saveDAT==1) 
		{
			printf("Saving %s_z=%ld_%i.dat...", FILE_DAT_BASENAME, hNz, q);
			if((io_error = save_dat(FILE_DAT_BASENAME, cFinif, q)) != 0)
			{
				print_error("Error: saving .dat file", io_error);
			}
			else
			{
				printf( "OK \n");
			}
			
			if (SaveEnergy==1)
			{
				printf("Saving ENERGY %s_z=%ld_%i.dat...", FILE_E_BASENAME, hNz, q);
				if((io_error = save_dat(FILE_E_BASENAME, Energia, q)) != 0)
				{
					print_error("Error: saving .dat file", io_error);
				}
				else
				{
					printf( "OK \n");
				}
			}
		}
			
		if(k == hNF-1)
		{
			printf("Saving E TOTAL %s_z=%ld_%i.dat...", FILE_PHI_TOTAL_DAT_BASENAME, hNz, q);
			if((io_error = save_dat_Energy(FILE_PHI_TOTAL_DAT_BASENAME, FinalEnergy, FinalPhi, q)) != 0)
			{
				print_error("Error: saving .dat file", io_error);
			}
			else
			{
				printf( "OK \n");
			}
			
			printf("Saving ENERGY LAYERS %s_z=%ld_%i.dat...", FILE_PHI_DAT_BASENAME_CAPAS, hNz, q);
			if((io_error = save_dat_capas(FILE_PHI_DAT_BASENAME_CAPAS, EnergiaTotal, FinalPhi2, q)) != 0)
			{
				print_error("Error: saving .dat file", io_error);
			}
			else
			{
				printf( "OK \n");
			}
				
		}
		
		if (savePLT==1) 
		{
			printf("Saving %s_z=%ld_%i.plt...", FILE_PLT_BASENAME, hNz, q);
			if((io_error = save_plt(FILE_PLT_BASENAME, cFinif, q)) != 0)
			print_error("Error: saving .plt file", io_error);
			else
			{
				printf( "OK \n");
			}
			
			if (SaveEnergy==1)
			{
				printf("Saving ENERGY %s_z=%ld_%i.plt...", FILE_E_BASENAME, hNz, q);
				if((io_error = save_plt(FILE_E_BASENAME, Energia, q)) != 0)
				print_error("Error: saving .plt file", io_error);
				else
				{
					printf( "OK \n");
				}		
			}
		}
		
		printf("Saving data end. Step: %i\n", q);

	}

}


//************************************************************************************************************
//									FILE MANAGMENT
//*************************************************************************************************************

// GET INITIAL DATA
errno_t get_param(const char *fname)
{ 
	// read main parameters from the input file "input3D.dat"
 FILE *fp;
 char buf[100];

 errno_t error = fopen_s(&fp, fname,"r");

 if(error != 0)
	 return error;
 
  char *token;
  char  delims[] = " ,\t\n";
  char* context	 = NULL;
    
 // read all input parameters
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&hNx);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&hNy);
 
 //Read the hNZs
 //First the count
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%d ",&hNz_count);
 //If there is any hNz then read all
 if(hNz_count > 0)
 {
	hNzs = (long*) malloc(hNz_count*sizeof(long));
	for(int i = 0; i < hNz_count; i++)
	{
		fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ", &(hNzs[i]));
	}
 }
 else
 {
 return -1 ;
 }
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&hHold);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&hz_AB);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&z_AT);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&hNF);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&htau);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&htauC);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&htauP);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&ha);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hf);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hfP);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hv);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hu);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hD);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hDP);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hDx);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hDy);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hB);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&hBP);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&etaI);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ",&etaR);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&idum);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&Nsteps);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&NstepsClean);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ", &hDT);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ", &hWd);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ", &hLa);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ", &hLb);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%f ", &hAmpl);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&hNpot);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&Vini);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%ld ",&Metodo);

 fgets(buf,100,fp); 
 token = strtok_s(buf, delims, &context); 
 sscanf(token,"%s", ext);

 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%d ", &saveDAT);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%d ", &savePLT);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%d ", &SaveEnergy);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%d ", &Nthread);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%s", FILE_RAND_BASENAME);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%s", FILE_PLT_BASENAME);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%s", FILE_DAT_BASENAME); 
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%s", FILE_E_BASENAME);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%s", FILE_PHI_TOTAL_DAT_BASENAME);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%s", FILE_PHI_DAT_BASENAME_CAPAS);
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%s", FILE_OPEN_BASENAME); 
 fgets(buf,100,fp);  token = strtok_s(buf, delims, &context); sscanf(token,"%s", FILE_OPEN_BASENAME_2); 
 
 fclose(fp); 


 //// print to check
 printf("Nx           = %ld, Ny=%ld NF=%ld\n", hNx, hNy, hNF);
 
 if(hNz_count > 0)
 {
	for(int i = 0; i < hNz_count; i++)
	{
 		 printf("hNz %i        = %ld \n", i, (hNzs[i]));
	}
 }
 printf("Hold            = %ld \n", hHold);
 printf("Hz_align bottom = %ld \n", hz_AB);
 printf("Hz_align top    = %ld \n", z_AT);
 printf("tau             = %f \n", htau);
 printf("tauC            = %f \n", htauC);
 printf("tauP            = %f \n", htauP);
 printf("a               = %f \n", ha);
 printf("f               = %f \n", hf);
 printf("fP              = %f \n", hfP);
 printf("v               = %f \n", hv);
 printf("u               = %f \n", hu);
 printf("D               = %f \n", hD);
 printf("DP              = %f \n", hDP);
 printf("Dx              = %f \n", hDx);
 printf("Dy              = %f \n", hDy);
 printf("B               = %f \n", hB);
 printf("BP              = %f \n", hBP);
 printf("etaI            = %f \n", etaI);
 printf("etaR            = %f \n", etaR);
 printf("idum            = %ld \n", idum);
 printf("Nsteps          = %ld \n", Nsteps);
 printf("NstepsClean     = %ld \n", NstepsClean);
 printf("DT              = %f \n", hDT);
 printf("Wd              = %f \n", hWd);
 printf("La              = %f \n", hLa);
 printf("Lb              = %f \n", hLb);
 printf("Ampl            = %f \n", hAmpl);
 printf("Npot            = %ld \n", hNpot);
 printf("Vini            = %ld \n", Vini);
 printf("Method          = %ld \n", Metodo);
 printf("Save phi.dat    = %i \n", saveDAT);
 printf("Save phi.plt    = %i \n", savePLT);
 printf("Save E.dat.plt  = %i \n", SaveEnergy); 
 printf("Extension       = %s \n", ext);
 printf("Nthread         = %d \n", Nthread);
 printf("RandName        = %s  \n", FILE_RAND_BASENAME);
 printf("PLT Name        = %s  \n", FILE_PLT_BASENAME);
 printf("DAT Name        = %s  \n", FILE_DAT_BASENAME);
 printf("Energy Name     = %s  \n", FILE_E_BASENAME);
 printf("<Phi><E>vs t    = %s  \n", FILE_PHI_TOTAL_DAT_BASENAME);
 printf("<Phi><E>vs layer= %s  \n", FILE_PHI_DAT_BASENAME_CAPAS); 
 printf("OPEN Name 1     = %s  \n", FILE_OPEN_BASENAME);
 printf("OPEN Name 2     = %s  \n", FILE_OPEN_BASENAME_2);
 fclose(fp);  
 return(0);
}

//OPEN A FILE .DAT
errno_t get_dat(char *fname)
{
	FILE *fp;

	errno_t error = fopen_s(&fp, fname,"r");
	if(error != 0)
		return error;
 
	for (int i=0; i < Ntotal; i++) 
	{
		fscanf(fp, "%f", &Fini[i]);
	}

	fclose(fp);
	return 0;
}

//OPEN A FILE .PLT
errno_t get_plt(const char *fname)
{
 FILE *fp;
 char buf[100];
 double val0, val1, val2;

 errno_t error = fopen_s(&fp, fname,"r");
 if(error != 0) 
	 return error;
 
 //Dismissing the 3 first lines
 fgets(buf,100,fp);
 fgets(buf,100,fp);
 fgets(buf,100,fp);

 	for (int i = 0; i < Ntotal; i++) {
		fscanf(fp, "%f %f %f %f", &val0, &val1, &val2, &Fini[i]); 
		//printf("Fini[%d]=%f\n",i,Fini[i]); if (i%10==0) {getchar();}
	}

 fclose(fp);

 return 0;
}

errno_t get_plt_Alineado_BOTTOM(int hz_AB, const char *fname)
{
 FILE *fp;
 char buf[100];
 double val0, val1, val2;

 int offset = hz_AB * hNx *hNy;

 errno_t error = fopen_s(&fp, fname,"r");
 if(error != 0) 
	 return error;
 
 //Dismissing the 3 first lines
 fgets(buf,100,fp);
 fgets(buf,100,fp);
 fgets(buf,100,fp);

	if (hNpot == 1) 
	{   int ipot = hLb + (hWd/2);
		for (int i = ipot ; i < (offset + ipot); i++)
		//for (int i = 0 ; i < (offset); i++) 
		{//desde comienzo del potencial
		fscanf(fp, "%f %f %f %f", &val0, &val1, &val2, &Fini[i]); 
		//printf("Fini[%d]=%f\n",i,Fini[i]); if (i%10==0) {getchar();}
		}
	}

	else
	{
		for (int i = 0 ; i < offset; i++) 
		{//desde comienzo del potencial
		fscanf(fp, "%f %f %f %f", &val0, &val1, &val2, &Fini[i]); 
		//printf("Fini[%d]=%f\n",i,Fini[i]); if (i%10==0) {getchar();}
		}
	}

 fclose(fp);

 return 0;
}

errno_t get_plt_Alineado_TOP(int hz_AB, const char *fname)
{
 FILE *fp;
 char buf[100];
 double val0, val1, val2;

 int offset = hz_AB * hNx *hNy;
 
 errno_t error = fopen_s(&fp, fname,"r");
 if(error != 0) 
	 return error;
 
 //Dismissing the 3 first lines
 fgets(buf,100,fp);
 fgets(buf,100,fp);
 fgets(buf,100,fp);

if (hNpot == 1) 
	{   int ipot = hLb + (hWd/2);
		for (int i = offset + ipot; i < Ntotal; i++) 
		//for (int i = offset; i < Ntotal; i++) 
		{
 		fscanf(fp, "%f %f %f %f", &val0, &val1, &val2, &Fini[i]); 
		}
	}

 else
	{
	 for (int i = offset; i < Ntotal; i++) 
	 {
 		fscanf(fp, "%f %f %f %f", &val0, &val1, &val2, &Fini[i]); 
 		//printf("Fini[%d]=%f\n",i,Fini[i]); if (i%10==0) {getchar();}
	}
	}

 fclose(fp);

 return 0;
}

//SAVE A .DAT
errno_t save_dat(const char *basename, float *cFini, int q)
{	FILE *fp;
	char filename[FILENAME_MAX];
	char cero[10];
	
	if (q < 10) 
		sprintf(cero, "0");
	else 
		sprintf(cero, "");
		
	sprintf_s(filename, "%s_z=%ld_%s%d.dat", basename, hNz, cero, q);
 
	errno_t error = fopen_s(&fp, filename,"w+");
	if(error != 0)
		return error;
 
	 	for (int i = 0; i < Ntotal; i++) 
		{
			fprintf(fp, "%11.6f\n", cFini[i]);
		}

	fclose(fp);

	return 0;
}


// SAVE A .PLT
errno_t save_plt(const char *basename, float *cFini, int q)
{
 FILE *fp;
 char filename[FILENAME_MAX];
 char cero[10];
	
	if (q < 10) 
		sprintf(cero, "0");
	else 
		sprintf(cero, "");
		
 sprintf_s(filename, "%s_z=%ld_%s%d.plt", basename, hNz, cero, q);

 errno_t error = fopen_s(&fp, filename, "w+");
 if(error != 0)
	 return error;

 char header1[100];
 char header2[100];
 char header3[100];

 sprintf_s(header1, "TITLE =\"Pattern 2D\"");
 sprintf_s(header2, "VARIABLES = \"X\",\"Y\",\"Z\",\"Psi\"");
 sprintf_s(header3, "ZONE I=%d, J=%d, K=%d, F=Point", hNx, hNy, hNz);
 
 fprintf(fp, "%s\n", header1);
 fprintf(fp, "%s\n", header2);
 fprintf(fp, "%s\n", header3);
 
 int Nxy = hNx * hNy;
 
 for(int k = 0; k < hNz; k++){
	for(int i = 0; i < hNy; i++){
		for(int j = 0; j < hNx; j++){

			fprintf(fp, "%3d.00 %3d.00 %3d.000000 %1.6f\n", i, j, k, cFini[k * Nxy + j * hNy + i]);
		}
	}
}
	
 fclose(fp);

 return 0;
}

// SAVE A ENERGY VS ESPESOR
errno_t save_dat_Energy2(const char *basename, float EnergiaTotal, float FinalPhi, int k, int q)
{	
	FILE *fp;
	char filename[FILENAME_MAX];
	char cero[10];
	
	if (q < 10) 
		sprintf(cero, "0");
	else 
		sprintf(cero, "");
		
	sprintf_s(filename, "%s_z=%ld_all.dat", basename, hNz);
	
	errno_t error = fopen_s(&fp, filename,"a");
	
	
	if(error != 0)
		return error;
		
			fprintf(fp, "%ld %ld %11.6f %11.6f\n", hNz, k, EnergiaTotal, FinalPhi);

	fclose(fp);

	return 0;
}

// SAVE A ENERGY VS ESPESOR
errno_t save_dat_Energy(const char *basename, float EnergiaTotal, float FinalPhi, int q)
{	
	FILE *fp;
	char filename[FILENAME_MAX];
	char cero[10];
	
	if (q < 10) 
		sprintf(cero, "0");
	else 
		sprintf(cero, "");
		
	sprintf_s(filename, "%s.dat", basename);
 
		
	errno_t error = fopen_s(&fp, filename,"a");
	
	
	if(error != 0)
		return error;
		
			fprintf(fp, "%ld %11.6f %11.6f\n", hNz, EnergiaTotal, FinalPhi);

	fclose(fp);

	return 0;
}

//SAVE A .DAT ENERGY POR CAPAS
errno_t save_dat_capas(const char *basename, float *energy, float *phi, int q)
{	
	FILE *fp;
	char filename[FILENAME_MAX];
	char cero[10];
	
	if (q < 10) 
		sprintf(cero, "0");
	else 
		sprintf(cero, "");
		
	sprintf_s(filename, "%s_z=%ld_%s%d.dat", basename, hNz, cero, q);
 
	errno_t error = fopen_s(&fp, filename,"w+");
	
	if(error != 0)
		return error;
 
	float Nxy = hNx*hNy;
	for(int j = 0; j < hNz; j++)
	{
		fprintf(fp, "%d %11.6f %11.6f\n", j, energy[j]/(Nxy), phi[j]);
	}
		

	fclose(fp);

	return 0;
}


//GENERATE RANDOM MATRIX FINI 
errno_t gen_rand(const char* fname, float *Fini){
 FILE *fp;
 
	char filename[FILENAME_MAX];
	sprintf_s(filename, "%s.dat", fname);

	errno_t error = fopen_s(&fp, filename, "w+");

	if(error != 0)
		return error;
 
	for (int i=0; i < Ntotal; i++) 
	{
		Fini[i] = etaI*(1.0f - 2.0f * ran1(&idum));
		fprintf(fp, "%15.6E\n", Fini[i]);
	}
	
 fclose(fp);
 return 0;
}

errno_t gen_rand_Alineado_BOTTOM(int hz_AB, const char* fname, float *Fini){
 FILE *fp;
 
	char filename[FILENAME_MAX];
	sprintf_s(filename, "%s_alineado.dat", fname);

	errno_t error = fopen_s(&fp, filename, "w+");

	if(error != 0)
		return error; 
 
	int offset = hz_AB * hNx * hNy;
 
 	for (int i = offset; i < Ntotal; i++) 
	{
		Fini[i] = etaI*(1.0f - 2.0f * ran1(&idum));
		fprintf(fp, "%15.6E\n", Fini[i]);
	}
	
 fclose(fp);
 return 0;
}

//PRINT ERROR
void print_error(const char *msg, int err) 
{
	printf("%s .Errno %i\n", msg, err);
	printf("Press any key to exit\n");
	char buf[10];
	scanf_s(buf);
	exit(err);
}

//WAIT BEFORE EXIT
void waitKey()
{
	printf("Press any key to continue...\n");
	char buf[10];
	scanf_s(buf);
}